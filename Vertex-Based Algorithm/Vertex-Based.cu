/*************************************************************************************************************************************************
Implementing Single Source Shortest Path given in TTCS paper "Locality-Based Relaxation: An Efficient Method for GPU-Based Computation of Shortest Paths", 2017.

Created by Mohsen Safari.
**************************************************************************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <cutil.h>

#define MAX_THREADS_PER_BLOCK 256
#define MAX_COST 10000000

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//CUDA Kernels
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void DijkastraKernel1(int* g_graph_nodes, int* g_graph_edges,int* g_graph_weights, bool* g_graph_mask1, bool* g_graph_mask2 , int* g_cost , int no_of_nodes, int edge_list_size, bool *d_finished)
{
	int tid = (blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);	
	int tid1 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);
	int tid2 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x)+1;
	if(tid == (no_of_nodes/2))
		tid2 = no_of_nodes - 1;
	if(tid2<no_of_nodes)
	{
		int i, j, k, l, edge_reg_i, edge_reg_j, edge_reg_k, edge_reg_l, end, end1, end2, end3;
		end = end1 = end2 = end3 = edge_list_size;
		if(g_graph_mask1[tid1])
		{
			g_graph_mask1[tid1] = false;
			if(tid1 != no_of_nodes-1)
				end = g_graph_nodes[tid1+1];
			for(i = g_graph_nodes[tid1]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
       				if(g_cost[tid1]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
				
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid1]+g_graph_weights[i]);
					if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);	
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask2[edge_reg_l] = true;
        	  									*d_finished = true;	
										}	
									}
								}
							}
						}
					}
          			
			
	    			}
			}
		}
		if(g_graph_mask1[tid2])
		{	
			g_graph_mask1[tid2] = false;
			end = g_graph_nodes[tid2+1];
			if(tid2 == no_of_nodes-1)
				end = edge_list_size;	
			for(i = g_graph_nodes[tid2]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
        			if(g_cost[tid2]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid2]+g_graph_weights[i]);
					end1 = edge_list_size;
          				if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							end2 = edge_list_size;
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);
									end3 = edge_list_size;
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask2[edge_reg_l] = true;
        	  									*d_finished = true;	
										}	
									}
								}
							}	
						}
					}
				
				}
	    		} 
	

		}
			
 	 }

}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void DijkastraKernel2(int* g_graph_nodes, int* g_graph_edges,int* g_graph_weights, bool* g_graph_mask1, bool* g_graph_mask2 , int* g_cost , int no_of_nodes, int edge_list_size, bool *d_finished)
{
	int tid = (blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);	
	int tid1 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);
	int tid2 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x)+1;
	if(tid == (no_of_nodes/2))
		tid2 = no_of_nodes - 1;
	if(tid2<no_of_nodes)
	{
		int i, j, k, l, edge_reg_i, edge_reg_j, edge_reg_k, edge_reg_l, end, end1, end2, end3;
		end = end1 = end2 = end3 = edge_list_size;
		if(g_graph_mask2[tid1])
		{
			g_graph_mask2[tid1] = false;
			if(tid1 != no_of_nodes-1)
				end = g_graph_nodes[tid1+1];
			for(i = g_graph_nodes[tid1]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
        			if(g_cost[tid1]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
				
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid1]+g_graph_weights[i]);
					if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);	
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask1[edge_reg_l] = true;
        	  									*d_finished = true;	
										}	
									}	
								}
							}
						}
					}
          			
			
	    			}
			}
		}
		if(g_graph_mask2[tid2])
		{	
			g_graph_mask2[tid2] = false;
			end = g_graph_nodes[tid2+1];
			if(tid2 == no_of_nodes-1)
				end = edge_list_size;	
			for(i = g_graph_nodes[tid2]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
        			if(g_cost[tid2]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid2]+g_graph_weights[i]);
					end1 = edge_list_size;
          				if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							end2 = edge_list_size;
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);
									end3 = edge_list_size;
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask1[edge_reg_l] = true;
        	  									*d_finished = true;	
										}	
									}
								}
							}	
						}
					}
				}
	    		}
		}
			
 	 }

}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void DijkastraKernel3(int* g_graph_nodes, int* g_graph_edges,int* g_graph_weights, bool* g_graph_mask1, bool* g_graph_mask2 , int* g_cost , int no_of_nodes, int edge_list_size, bool *d_finished)
{
	int tid = (blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);	
	int tid1 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);
	int tid2 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x)+1;
	if(tid == (no_of_nodes/2))
		tid2 = no_of_nodes - 1;
	if(tid2<no_of_nodes)
	{
		int i, j, k, l, edge_reg_i, edge_reg_j, edge_reg_k, edge_reg_l, end, end1, end2, end3;
		end = end1 = end2 = end3 = edge_list_size;
		if(g_graph_mask1[tid1])
		{
			g_graph_mask1[tid1] = false;
			if(tid1 != no_of_nodes-1)
				end = g_graph_nodes[tid1+1];
			for(i = g_graph_nodes[tid1]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
        			if(g_cost[tid1]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
				
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid1]+g_graph_weights[i]);
					if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);	
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask2[edge_reg_l] = true;
										}	
									}
								}
							}
						}
					}
          			
			
	    			}
			}
		}
		if(g_graph_mask1[tid2])
		{	
			g_graph_mask1[tid2] = false;
			end = g_graph_nodes[tid2+1];
			if(tid2 == no_of_nodes-1)
				end = edge_list_size;	
			for(i = g_graph_nodes[tid2]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
        			if(g_cost[tid2]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid2]+g_graph_weights[i]);
					end1 = edge_list_size;
          				if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							end2 = edge_list_size;
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);
									end3 = edge_list_size;
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask2[edge_reg_l] = true;	
										}	
									}
								}
							}	
						}
					}
				}
	    		}
		}
			
  	}

}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void DijkastraKernel4(int* g_graph_nodes, int* g_graph_edges,int* g_graph_weights, bool* g_graph_mask1, bool* g_graph_mask2 , int* g_cost , int no_of_nodes, int edge_list_size, bool *d_finished)
{
	int tid = (blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);	
	int tid1 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x);
	int tid2 = 2*(blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x)+1;
	if(tid == (no_of_nodes/2))
		tid2 = no_of_nodes - 1;
	if(tid2<no_of_nodes)
	{
		int i, j, k, l, edge_reg_i, edge_reg_j, edge_reg_k, edge_reg_l, end, end1, end2, end3;
		end = end1 = end2 = end3 = edge_list_size;
		if(g_graph_mask2[tid1])
		{
			g_graph_mask2[tid1] = false;
			if(tid1 != no_of_nodes-1)
				end = g_graph_nodes[tid1+1];
			for(i = g_graph_nodes[tid1]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
        			if(g_cost[tid1]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
				
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid1]+g_graph_weights[i]);
					if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);	
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask1[edge_reg_l] = true;	
										}	
									}		
								}
							}
						}
					}
          			
			
	    			}
			}
		}
		if(g_graph_mask2[tid2])
		{	
			g_graph_mask2[tid2] = false;
			end = g_graph_nodes[tid2+1];
			if(tid2 == no_of_nodes-1)
				end = edge_list_size;	
			for(i = g_graph_nodes[tid2]; i<end; i++)
			{
				edge_reg_i = g_graph_edges[i];
        			if(g_cost[tid2]+g_graph_weights[i] < g_cost[edge_reg_i])
        			{
	  				atomicMin(&g_cost[edge_reg_i], g_cost[tid2]+g_graph_weights[i]);
					
          				if (g_graph_nodes[i] != no_of_nodes-1)
						end1 = g_graph_nodes[edge_reg_i+1];
					for(j = g_graph_nodes[edge_reg_i]; j<end1; j++)
					{
						edge_reg_j = g_graph_edges[j];
						if(g_cost[edge_reg_i]+g_graph_weights[j] < g_cost[edge_reg_j])
						{
							atomicMin(&g_cost[edge_reg_j], g_cost[edge_reg_i]+g_graph_weights[j]);
							end2 = edge_list_size;
							if (g_graph_nodes[j] != no_of_nodes-1)
								end2 = g_graph_nodes[edge_reg_j+1];
							for(k = g_graph_nodes[edge_reg_j]; k<end2; k++)
							{
								edge_reg_k = g_graph_edges[k];
								if(g_cost[edge_reg_j]+g_graph_weights[k] < g_cost[edge_reg_k])
								{
									atomicMin(&g_cost[edge_reg_k], g_cost[edge_reg_j]+g_graph_weights[k]);
									end3 = edge_list_size;
									if (g_graph_nodes[k] != no_of_nodes-1)
										end3 = g_graph_nodes[edge_reg_k+1];
									for(l = g_graph_nodes[edge_reg_k]; l<end3; l++)
									{
										edge_reg_l = g_graph_edges[l];
										if(g_cost[edge_reg_k]+g_graph_weights[l] < g_cost[edge_reg_l])
										{
											atomicMin(&g_cost[edge_reg_l], g_cost[edge_reg_k]+g_graph_weights[l]);	
											g_graph_mask1[edge_reg_l] = true;	
										}	
									}	
								}
							}	
						}
					}
				}
	    		}
		}
			
  	}

}

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	int repetition = 100;  // Repeat the algorithm for this number (of random sources) and take average time  
	float total_time=0;
	int query, queryarray[repetition];
	FILE *fp0;
	fp0 = fopen("/home/mohsen/Input/NewYorkQueries.txt", "r"); // The address of a query file to get sources from the input graph
	if(!fp0)
	{
		printf("Error reading query file\n");
		return 0;
	}
	for( unsigned int i = 0; i < repetition; i++) // Store the random sources from the query file
   	 {
		fscanf(fp0,"%d",&query);
        	queryarray[i] = query;
   	 }
	
	fclose(fp0); 
	
	for(int r=0; r<repetition; r++) // Repeat the program for a fix number of random sources and at the end take an average over time
	{	
		int no_of_nodes = 0;
		int edge_list_size = 0;
		FILE *fp;
		
		fp = fopen("/home/mohsen/Input/NewYork-CSR.txt", "r"); // The address of the input graph
		if(!fp)
		{
			printf("Error reading graph file\n");
			return 0;
		}
	
		int source = 0;
		
		fscanf(fp,"%d",&no_of_nodes);
		printf("Number of nodes: %d\n ",no_of_nodes);
	
		int num_of_blocks = 1;
		int num_of_threads_per_block = no_of_nodes;
	
		
		
		if(no_of_nodes>MAX_THREADS_PER_BLOCK) // Distribute threads across multiple blocks if necessary
		{
			num_of_blocks = (int)ceil(no_of_nodes/(double)(2*MAX_THREADS_PER_BLOCK)); 
			num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
		}
		
		// Allocate Host memory
    		int* h_graph_nodes = (int*) malloc(sizeof(int)*no_of_nodes);
    		bool *h_graph_mask1 = (bool*) malloc(sizeof(bool)*no_of_nodes);
    		bool *h_graph_mask2 = (bool*) malloc(sizeof(bool)*no_of_nodes);

    		int start, edgeno;   

   		// Initalize the memory
		int no = 0;
   		for( unsigned int i = 0; i < no_of_nodes; i++) 
   		{
			fscanf(fp,"%d %d",&start,&edgeno);
			if(edgeno>100)
				no++;
        		h_graph_nodes[i] = start;
        		h_graph_mask1[i] = false;
			h_graph_mask2[i] = false;
    		}
    
    		// Read the first source from the file
   		fscanf(fp,"%d",&source);
        
      		// Read and store edges and weights
    		fscanf(fp,"%d",&edge_list_size);
    		printf("Number of edges: %d\n",edge_list_size);    
    		int id;
    		int* h_graph_edges = (int*) malloc(sizeof(int)*edge_list_size);
    		int* h_graph_weights = (int*) malloc(sizeof(int)*edge_list_size);
    		for(int i=0; i < edge_list_size ; i++)
    		{
			fscanf(fp,"%d",&id);
			h_graph_edges[i] = id;
			fscanf(fp,"%d",&id);
			h_graph_weights[i] = id;
		
    		}
    
    
		if(fp)
			fclose(fp);    


		// Allocate and initialize the memory 
		int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
		for(int i=0;i<no_of_nodes;i++)
			h_cost[i]= MAX_COST;	
	
		// Initialize the source
		source = queryarray[r]; 
        	h_cost[source] = 0;
        	h_graph_mask1[source] = true;
	
	
		// Copy arrays from Host to Device memory
    		int* d_graph_nodes;
    		hipMalloc( (void**) &d_graph_nodes, sizeof(int)*no_of_nodes) ;
    		hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;

		int* d_graph_edges;
    		hipMalloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size) ;
    		hipMemcpy( d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;

		int* d_graph_weights;
    		hipMalloc( (void**) &d_graph_weights, sizeof(int)*edge_list_size) ;
    		hipMemcpy( d_graph_weights, h_graph_weights, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;
    
    		bool* d_graph_mask1;
    		hipMalloc( (void**) &d_graph_mask1, sizeof(bool)*no_of_nodes) ;
    		hipMemcpy( d_graph_mask1, h_graph_mask1, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;

		bool* d_graph_mask2;
    		hipMalloc( (void**) &d_graph_mask2, sizeof(bool)*no_of_nodes) ;
    		hipMemcpy( d_graph_mask2, h_graph_mask2, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
    
    		int* d_cost;
    		hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes);
    		hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;

	
    		// A boolean to check the termination of the algorithm
		bool *d_finished;
		bool finished;
		hipMalloc( (void**) &d_finished, sizeof(bool));
    
        	// Setup execution parameters
        	dim3  grid( num_of_blocks, 1, 1);
        	dim3  threads( num_of_threads_per_block, 1, 1);

		int* temp = (int*) malloc(sizeof(int)*no_of_nodes);
		int* SSSP = (int *)malloc((no_of_nodes+1) * sizeof(int *));
		int counter = 0;

		// Start the timer
  		hipEvent_t begin, end;
		float time;
		hipEventCreate(&begin);
		hipEventCreate(&end);
		hipEventRecord(begin, 0);

		// Kernel launches without CPU-GPU communication for a fix number (counter) for each graph
		do
		{
			DijkastraKernel3<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_weights, d_graph_mask1, d_graph_mask2,    				d_cost, no_of_nodes, edge_list_size, d_finished);
			DijkastraKernel4<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_weights, d_graph_mask1, d_graph_mask2, 				d_cost, no_of_nodes, edge_list_size, d_finished);
			counter++;
		
		}
		while(counter<=64);

		// Kernel launches with CPU-GPU communication via a boolean variable (finished)
		do
		{
			finished=false;
			hipMemcpy( d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice) ;
			DijkastraKernel1<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_weights, d_graph_mask1, d_graph_mask2, 				d_cost, no_of_nodes, edge_list_size, d_finished);
			DijkastraKernel2<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_weights, d_graph_mask1, d_graph_mask2, 				d_cost, no_of_nodes, edge_list_size, d_finished);
			finished=false;
			hipMemcpy( &finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost) ;
		
		}
		while(finished);
    
    
    		// Copy result from Device to host
   		hipMemcpy( temp, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) ;

		// Store results
		SSSP[0] = source;			
		for(int p=1;p<=no_of_nodes;p++)
			SSSP[p] = temp[p-1];	

		// Stop the timer
    		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&time, begin, end);
	
		// Record the time for each repetition
		total_time+=time;

		hipEventDestroy(begin);
		hipEventDestroy(end);
	
	
		// Store the result into a file
		//FILE *fpo = fopen("/home/mohsen/result-alg35.txt","w");
		//for(int j=1;j<=no_of_nodes;j++)
		//	fprintf(fpo, "%d  %d  %d\n", SSSP[0], j-1, SSSP[j]);
		//fclose(fpo);
		//printf("Results stored in result of SSSP(Harish).txt\n");
	
	
    		// Cleanup memory
   		free( h_graph_nodes);
   		free( h_graph_edges);
   		free( h_graph_mask1);
    		free( h_graph_weights);
		free( h_graph_mask2);
    		free( h_cost);
		free(temp);
		free(SSSP);
    		hipFree(d_graph_nodes);
    		hipFree(d_graph_edges);
    		hipFree(d_graph_mask1);
    		hipFree(d_graph_weights);
		hipFree(d_graph_mask2);
    		hipFree(d_cost);
		hipFree(d_finished);	

	}

	// Take an average over total time
	printf( "Processing time: %f (ms)\n", (float)(total_time/repetition));

	return 0;
       
}

